#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void threed_mem_transfer(int* input, int size)
{
	int tidx = threadIdx.x;
	int tidy = blockDim.x*threadIdx.y;
	int tidz = blockDim.x* blockDim.x*threadIdx.z;
	int xblock_index = blockDim.x * blockIdx.x;
	int row_index = blockDim.x * blockDim.y * gridDim.x * blockIdx.y;
	int page_index = blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * blockIdx.z;
	int gid = tidx + tidy + tidz + row_index + page_index;
	if (gid < size) {
		printf("gid: %d, value: %d \n", gid, input[gid]);
	}
}

int main() 
{
	int size = 64;
	int byte_size = size * sizeof(int);

	int* h_input;
	h_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned) time(&t));
	for (int i = 0; i < size; i++) {
		h_input[i] = (int)(rand() & 0xff);
	}

	int* d_input;
	hipMalloc((void**)& d_input, byte_size);

	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	dim3 block(2,2,2);
	dim3 grid(2,2,2);

	threed_mem_transfer << <grid, block >> > (d_input, size);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}