#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

/*__global__ void addkernel(int *c, const int *a, const int *b)
{
    int i = threadidx.x;
    c[i] = a[i] + b[i];
}*/

__global__ void hellow_cuda() {
	printf("hello CUDA world \n");
}

int main()
{

	/*hellow_cuda << <2, 5 >> > ();*/
	int nx, ny;
	nx = 16;
	ny = 4;

	dim3 block(8,2,1);
	dim3 grid(nx/block.x, ny/block.y);

	hellow_cuda << <grid, block >> > ();

	hipDeviceSynchronize();

	hipDeviceReset();

    return 0;
}